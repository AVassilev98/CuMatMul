#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <stdint.h>
#include <hipblas.h>
#include <hiprand.h>
#include <string>
#include <functional>
#include "matMul.h"

#ifdef CUBLAS_API_H_
static const char *_cublasGetError(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";

        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "HIPBLAS_STATUS_NOT_SUPPORTED";

        case HIPBLAS_STATUS_UNKNOWN:
            return "HIPBLAS_STATUS_UNKNOWN";
    }

    return "<unknown>";
}
#endif

#ifdef CURAND_H_
static const char *_curandGetError(hiprandStatus_t error)
{
    switch (error)
    {
        case HIPRAND_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPRAND_STATUS_VERSION_MISMATCH:
            return "HIPRAND_STATUS_VERSION_MISMATCH";

        case HIPRAND_STATUS_NOT_INITIALIZED:
            return "HIPRAND_STATUS_NOT_INITIALIZED";

        case HIPRAND_STATUS_ALLOCATION_FAILED:
            return "HIPRAND_STATUS_ALLOCATION_FAILED";

        case HIPRAND_STATUS_TYPE_ERROR:
            return "HIPRAND_STATUS_TYPE_ERROR";

        case HIPRAND_STATUS_OUT_OF_RANGE:
            return "HIPRAND_STATUS_OUT_OF_RANGE";

        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
            return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";

        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
            return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";

        case HIPRAND_STATUS_LAUNCH_FAILURE:
            return "HIPRAND_STATUS_LAUNCH_FAILURE";

        case HIPRAND_STATUS_PREEXISTING_FAILURE:
            return "HIPRAND_STATUS_PREEXISTING_FAILURE";

        case HIPRAND_STATUS_INITIALIZATION_FAILED:
            return "HIPRAND_STATUS_INITIALIZATION_FAILED";

        case HIPRAND_STATUS_ARCH_MISMATCH:
            return "HIPRAND_STATUS_ARCH_MISMATCH";

        case HIPRAND_STATUS_INTERNAL_ERROR:
            return "HIPRAND_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
#endif

#define CUDA_CHECK_ERROR(expr)                                                                                          \
expr;                                                                                                                   \
{                                                                                                                       \
    hipError_t err = hipGetLastError();                                                                               \
    if (err != 0)                                                                                                       \
    {                                                                                                                   \
        fprintf(stderr, "%s:%d - %s failed with cuda error: %s\n", __FILE__, __LINE__, #expr, hipGetErrorString(err)); \
    }                                                                                                                   \
}

#define CUBLAS_CHECK_ERROR(expr)                                                                                            \
{                                                                                                                           \
    hipblasStatus_t status = expr;                                                                                           \
    if (status != HIPBLAS_STATUS_SUCCESS)                                                                                    \
    {                                                                                                                       \
        fprintf(stderr, "%s:%d - %s failed with cublas error: %s\n", __FILE__, __LINE__, #expr, _cublasGetError(status));   \
    }                                                                                                                       \
}

#define CURAND_CHECK_ERROR(expr)                                                                                            \
{                                                                                                                           \
    hiprandStatus_t status = expr;                                                                                           \
    if (status != HIPRAND_STATUS_SUCCESS)                                                                                    \
    {                                                                                                                       \
        fprintf(stderr, "%s:%d - %s failed with hiprand error: %s\n", __FILE__, __LINE__, #expr, _curandGetError(status));   \
    }                                                                                                                       \
}

#define CHECK_OR_PRINT(expr)                                                            \
{                                                                                       \
    if (!(expr))                                                                        \
    {                                                                                   \
        fprintf(stderr, "%s:%d - %s expression failed\n", __FILE__, __LINE__, #expr);   \
    }                                                                                   \
}

#define EPSILON 0.1f
static bool _validateMatMulResults(float *golden, float *mat, size_t dim)
{
    for (size_t i = 0; i < dim; i++)
    {
        for (size_t j = 0; j < dim; j++)
        {
            size_t idx = i * dim + j;
            if (fabs(golden[idx] - mat[idx]) > EPSILON)
            {
                fprintf(stderr, "row: %lu col: %lu -- expected: %f actual: %f\n", i, j, golden[idx], mat[idx]);
                return false;
            }
        }
    }

    return true;
}

#define DIV_CEIL(num, div) (((num) + (div) - 1) / (div))

struct CudaDeviceProperties
{
    int smVerMajor;
    int smVerMinor;
    int warpSize;
    int maxThreadsPerBlock;
    int maxBlockDimX;
    int maxBlockDimY;
    int maxGridDimX;
    int maxGridDimY;
    int maxSmemPerBlock;
    int totalConstMem;
    int maxRegistersPerBlock;
    int l2CacheSize;
    int smCount;
};
CudaDeviceProperties g_cudaDeviceProperties;

void cudaDeviceGetProperties(int deviceIdx)
{
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.smVerMajor, hipDeviceAttributeComputeCapabilityMajor, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.smVerMinor, hipDeviceAttributeComputeCapabilityMinor, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.warpSize, hipDeviceAttributeWarpSize, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.maxBlockDimX, hipDeviceAttributeMaxBlockDimX, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.maxBlockDimY, hipDeviceAttributeMaxBlockDimY, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.maxGridDimX, hipDeviceAttributeMaxGridDimX, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.maxGridDimY, hipDeviceAttributeMaxGridDimY, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.maxSmemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.totalConstMem, hipDeviceAttributeTotalConstantMemory, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.maxRegistersPerBlock, hipDeviceAttributeMaxRegistersPerBlock, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.l2CacheSize, hipDeviceAttributeL2CacheSize, deviceIdx));
    CUDA_CHECK_ERROR(hipDeviceGetAttribute(&g_cudaDeviceProperties.smCount, hipDeviceAttributeMultiprocessorCount, deviceIdx));

    printf("======= DEVICE %d ATTRIBUTES ======\n", deviceIdx);
    printf("smVerMajor:             %d\n", g_cudaDeviceProperties.smVerMajor);
    printf("smVerMinor:             %d\n", g_cudaDeviceProperties.smVerMinor);
    printf("warpSize:               %d\n", g_cudaDeviceProperties.warpSize);
    printf("maxThreadsPerBlock:     %d\n", g_cudaDeviceProperties.maxThreadsPerBlock);
    printf("maxBlockDimX:           %d\n", g_cudaDeviceProperties.maxBlockDimX);
    printf("maxBlockDimY:           %d\n", g_cudaDeviceProperties.maxBlockDimY);
    printf("maxGridDimX:            %d\n", g_cudaDeviceProperties.maxGridDimX);
    printf("maxGridDimY:            %d\n", g_cudaDeviceProperties.maxGridDimY);
    printf("maxSmemPerBlock:        %d\n", g_cudaDeviceProperties.maxSmemPerBlock);
    printf("totalConstMem:          %d\n", g_cudaDeviceProperties.totalConstMem);
    printf("maxRegistersPerBlock:   %d\n", g_cudaDeviceProperties.maxRegistersPerBlock);
    printf("l2CacheSize:            %d\n", g_cudaDeviceProperties.l2CacheSize);
    printf("smCount:                %d\n", g_cudaDeviceProperties.smCount);
    printf("======= DEVICE %d ATTRIBUTES ======\n\n", deviceIdx);
}

enum MatMulImplType
{
    MAT_MUL_IMPL_CUBLAS,
    MAT_MUL_IMPL_NAIVE,
    MAT_MUL_IMPL_COUNT,
};

struct MatMulImpl
{
    const std::string typeStr;
    std::function<void(float *, float *, float *, size_t)> implWrapper;
    float elapsedMs = 0.0f;
};

float runCuBLASMatMul
(
    float *d_matrixA,
    float *d_matrixB,
    float *d_matrixRes,
    size_t matDim
)
{
    float time = 0.0f;
    hipEvent_t start;
    hipEvent_t stop;
    CUDA_CHECK_ERROR(hipEventCreate(&start));
    CUDA_CHECK_ERROR(hipEventCreate(&stop));

    hipblasHandle_t handle;
    CUBLAS_CHECK_ERROR(hipblasCreate(&handle));

    static const float scaling = 1.0f;
    CUDA_CHECK_ERROR(hipEventRecord(start));
    CUBLAS_CHECK_ERROR(
        hipblasSgemm(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            matDim, matDim, matDim,
            &scaling,
            d_matrixA, matDim,
            d_matrixB, matDim,
            &scaling,
            d_matrixRes, matDim
        )
    );
    CUDA_CHECK_ERROR(hipEventRecord(stop));
    CUDA_CHECK_ERROR(hipEventSynchronize(stop));
    CUDA_CHECK_ERROR(hipEventElapsedTime(&time, start, stop));

    return time;
}

void runNaiveMatMul
(
    float *d_matrixA,
    float *d_matrixB,
    float *d_matrixRes,
    size_t matDim
)
{
    dim3 blockDim = {{
        .x = (uint32_t)g_cudaDeviceProperties.warpSize,
        .y = (uint32_t)(g_cudaDeviceProperties.maxThreadsPerBlock / g_cudaDeviceProperties.warpSize),
        .z = 1,
    }};

    dim3 gridDim = {{
        .x = (uint32_t)DIV_CEIL(matDim , blockDim.x),
        .y = (uint32_t)DIV_CEIL(matDim , blockDim.y),
        .z = 1,
    }};

    cuMatrixMulNaive<<<gridDim, blockDim>>>(d_matrixA, d_matrixB, d_matrixRes, matDim);
}

void runImplementations
(
    float *d_matrixA,
    float *d_matrixB,
    float *d_matrixRes,
    float *h_matrixRes,
    float *h_goldenResult,
    size_t matDim,
    std::array<MatMulImpl, MAT_MUL_IMPL_COUNT> &implArray
)
{
    hipEvent_t start;
    hipEvent_t stop;
    float cublasTime = 0.0f;
    CUDA_CHECK_ERROR(hipEventCreate(&start));
    CUDA_CHECK_ERROR(hipEventCreate(&stop));

    const size_t matSize = matDim * matDim;

    cublasTime = runCuBLASMatMul(d_matrixA, d_matrixB, d_matrixRes, matDim);
    hipMemcpy(h_goldenResult, d_matrixRes, matSize * sizeof(float), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < MAT_MUL_IMPL_COUNT; i++)
    {
        CUDA_CHECK_ERROR(hipMemset2D(d_matrixRes, matDim * sizeof(float), 0, matDim * sizeof(float), matDim))
        MatMulImpl &impl = implArray[i];
        CUDA_CHECK_ERROR(hipEventRecord(start));
        impl.implWrapper(d_matrixA, d_matrixB, d_matrixRes, matDim);
        CUDA_CHECK_ERROR(hipEventRecord(stop));
        CUDA_CHECK_ERROR(hipEventSynchronize(stop));
        CUDA_CHECK_ERROR(hipEventElapsedTime(&impl.elapsedMs, start, stop));

        hipMemcpy(h_matrixRes, d_matrixRes, matSize * sizeof(float), hipMemcpyDeviceToHost);

        if (_validateMatMulResults(h_goldenResult, h_matrixRes, matDim))
        {
            printf("Implementation: %s took %.1fms to execute, %.2fX as long as CUBLAS\n",
                impl.typeStr.c_str(),
                impl.elapsedMs,
                impl.elapsedMs / cublasTime);
        }
        else
        {
            printf("Errors detected in implementation: %s\n", impl.typeStr.c_str());
        }
    }
}

int main(int argc, char **argv)
{
    static const size_t N = 16384;
    static const size_t matSize = N * N;

    float *h_pMatrixRes = NULL;
    float *h_pMatrixResGolden = NULL;

    float *d_pMatrixA = NULL;
    float *d_pMatrixB = NULL;
    float *d_pMatrixRes = NULL;
    float *d_pMatrixResGolden = NULL;

    cudaDeviceGetProperties(0);

    CUDA_CHECK_ERROR(hipHostMalloc(&h_pMatrixRes, sizeof(float) * matSize));
    CUDA_CHECK_ERROR(hipHostMalloc(&h_pMatrixResGolden, sizeof(float) * matSize));

    CUDA_CHECK_ERROR(hipMalloc(&d_pMatrixA, sizeof(float) * matSize));
    CUDA_CHECK_ERROR(hipMalloc(&d_pMatrixB, sizeof(float) * matSize));
    CUDA_CHECK_ERROR(hipMalloc(&d_pMatrixRes, sizeof(float) * matSize));
    CUDA_CHECK_ERROR(hipMalloc(&d_pMatrixResGolden, sizeof(float) * matSize));

    hiprandGenerator_t generator;
    CURAND_CHECK_ERROR(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_XORWOW));
    // Don't need randomization across runs
    CURAND_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(generator, 0ULL));

    CURAND_CHECK_ERROR(hiprandGenerateUniform(generator, d_pMatrixA, matSize));
    CURAND_CHECK_ERROR(hiprandGenerateUniform(generator, d_pMatrixB, matSize));
    CUDA_CHECK_ERROR(hipMemset2D(d_pMatrixRes, N * sizeof(float), 0, N * sizeof(float), N))
    CUDA_CHECK_ERROR(hipMemset2D(d_pMatrixResGolden, N * sizeof(float), 0, N * sizeof(float), N))

    hiprandStatus_t hiprandGenerateNormal(
        hiprandGenerator_t generator, 
        float *outputPtr, size_t n, 
        float mean, float stddev);


    std::array<MatMulImpl, MAT_MUL_IMPL_COUNT> implArray = {{
        {.typeStr = "CUBLAS", .implWrapper = runCuBLASMatMul},
        {.typeStr = "Naive", .implWrapper = runNaiveMatMul},
    }};

    runImplementations(
        d_pMatrixA, 
        d_pMatrixB, 
        d_pMatrixRes, 
        h_pMatrixRes, 
        h_pMatrixResGolden, 
        N, 
        implArray);
}